#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a preconditioned conjugate gradient solver on
 * the GPU using CUBLAS and CUSPARSE.  Relative to the conjugateGradient
 * SDK example, this demonstrates the use of hipsparseScsrilu02() for
 * computing the incompute-LU preconditioner and hipsparseScsrsv2_solve()
 * for solving triangular systems.  Specifically, the preconditioned
 * conjugate gradient method with an incomplete LU preconditioner is
 * used to solve the Laplacian operator in 2D on a uniform mesh.
 *
 * Note that the code in this example and the specific matrices used here
 * were chosen to demonstrate the use of the CUSPARSE library as simply
 * and as clearly as possible.  This is not optimized code and the input
 * matrices have been chosen for simplicity rather than performance.
 * These should not be used either as a performance guide or for
 * benchmarking purposes.
 */

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipblas.h>
#include <hipsparse.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>       // CUDA error checking
#include <helper_functions.h>  // shared functions common to CUDA Samples

// #include "inc/kernel.cu"


const char *sSDKname = "conjugateGradientPrecond";


__global__
void saxpy(int n, float a, float *x, float *y, float *result)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) result[i] = a*x[i] + y[i];
}

__global__
void elementwise_mult(int n, float a, float *x, float *y, float *result)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) result[i] = a*x[i] * y[i];
}

/*
 * Generate a matrix representing a second order regular Laplacian operator
 * on a 2D domain in Compressed Sparse Row format.
 */
void genLaplace(int *row_ptr, int *col_ind, float *val, int M, int N, int nz,
                float *rhs, float *diag, int *end_ptr_even, int *end_ptr_odd,
                int *row_ptr_even, int *row_ptr_odd, int *idx_even, int *idx_odd) {
  assert(M == N);
  int n = (int)sqrt((double)N);
  assert(n * n == N);
  printf("laplace dimension = %d\n", n);
  int idx = 0;
  int idx_diag = 0;

  // loop over degrees of freedom
  for (int i = 0; i < N; i++) {
    int ix = i % n;
    int iy = i / n; 

    row_ptr[i] = idx;

    if ((ix + iy) % 2 == 0) {
      row_ptr_even[*idx_even] = idx;
//       *idx_even += 1;
    } else {
      row_ptr_odd[*idx_odd] = idx;
//       *idx_odd += 1;
    }

    // up
    if (iy > 0) {
      val[idx] = 1.0;
      col_ind[idx] = i - n;
      idx++;
//       if ((i + (i - n)) % 2 == 0) {
//         col_ind_even[*idx_even] = i - n;
//         *idx_even += 1;
//       } else {
//         col_ind_odd[*idx_odd] = i - n;
//         *idx_odd += 1;
//       }
    } else {
      rhs[i] -= 1.0;
    }

    // left
    if (ix > 0) {
      val[idx] = 1.0;
      col_ind[idx] = i - 1;
      idx++;
//       if ((i + (i - 1)) % 2 == 0) {
//         col_ind_even[*idx_even] = i - 1;
//         *idx_even += 1;
//       } else {
//         col_ind_odd[*idx_odd] = i - 1;
//         *idx_odd += 1;
//       }
    } else {
      rhs[i] -= 0.0;
    }

    // center
    val[idx] = -4.0;
//     val[idx] = 0.;
    diag[idx_diag] = 1. / (-4.);
    col_ind[idx] = i;
//     col_ind_even[*idx_even] = i;
    idx++;
//     *idx_even += 1;
    idx_diag++;

    // right
    if (ix < n - 1) {
      val[idx] = 1.0;
      col_ind[idx] = i + 1;
      idx++;
//       if ((i + (i + 1)) % 2 == 0) {
//         col_ind_even[*idx_even] = i + 1;
//         *idx_even += 1;
//       } else {
//         col_ind_odd[*idx_odd] = i + 1;
//         *idx_odd += 1;
//       }
    } else {
      rhs[i] -= 0.0;
    }

    // down
    if (iy < n - 1) {
      val[idx] = 1.0;
      col_ind[idx] = i + n;
      idx++;
//       if ((i + (i + n)) % 2 == 0) {
//         col_ind_even[*idx_even] = i + n;
//         *idx_even += 1;
//       } else {
//         col_ind_odd[*idx_odd] = i + n;
//         *idx_odd += 1;
//       }
    } else {
      rhs[i] -= 0.0;
    }

    if ((ix + iy) % 2 == 0) {
      end_ptr_even[*idx_even] = idx;
      *idx_even += 1;
    } else {
      end_ptr_odd[*idx_odd] = idx;
      *idx_odd += 1;
    }
  }


  row_ptr[N] = idx;
  printf("\n\n");
}

/*
 * Solve Ax=b using the conjugate gradient method
 * a) without any preconditioning,
 * b) using an Incomplete Cholesky preconditioner, and
 * c) using an ILU0 preconditioner.
 */
int main(int argc, char **argv) {
  const int max_iter = 1000;
  int k, M = 0, N = 0, nz = 0, *I = NULL, *J = NULL;
  int *J_even = NULL, *J_odd = NULL;
  int *I_even = NULL, *I_odd = NULL;
  int *d_col, *d_row;
  int qatest = 0;
  const float tol = 1e-12f;
  float *x, *rhs, *diag;
  float r0, r1, alpha, beta;
  float *d_val, *d_x;
  float *d_zm1, *d_zm2, *d_rm2;
  float *d_r, *d_p, *d_omega, *d_x_old, *d_diff;
  float *val = NULL;
  float *d_valsILU0;
  void *buffer = NULL;
  float rsum, diff, err = 0.0;
  float qaerr1, qaerr2 = 0.0;
  float dot, numerator, denominator, nalpha;
  const float floatone = 1.0;
  const float floatminusone = -1.0;
  const float floatzero = 0.0;
  int idx_even = 0, idx_odd = 0;
  float *d_diag;

  int nErrors = 0;

  printf("conjugateGradientPrecond starting...\n");

  /* QA testing mode */
  if (checkCmdLineFlag(argc, (const char **)argv, "qatest")) {
    qatest = 1;
  }

  /* This will pick the best possible CUDA capable device */
  hipDeviceProp_t deviceProp;
  int devID = findCudaDevice(argc, (const char **)argv);
  printf("GPU selected Device ID = %d \n", devID);

  if (devID < 0) {
    printf("Invalid GPU device %d selected,  exiting...\n", devID);
    exit(EXIT_SUCCESS);
  }

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  /* Statistics about the GPU device */
  printf(
      "> GPU device has %d Multi-Processors, "
      "SM %d.%d compute capabilities\n\n",
      deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

  /* Generate a Laplace matrix in CSR (Compressed Sparse Row) format */
//   M = N = 16384;
  M = N = 1024;
  nz = 5 * N - 4 * (int)sqrt((double)N);
  I = (int *)malloc(sizeof(int) * (N + 1));   // csr row pointers for matrix A
  J = (int *)malloc(sizeof(int) * nz);        // csr column indices for matrix A
  J_even = (int *)malloc(sizeof(int) * (N + 1));        // csr column indices for matrix A
  J_odd = (int *)malloc(sizeof(int) * (N + 1));        // csr column indices for matrix A
  I_even = (int *)malloc(sizeof(int) * (N + 1));        // csr column indices for matrix A
  I_odd = (int *)malloc(sizeof(int) * (N + 1));        // csr column indices for matrix A
  val = (float *)malloc(sizeof(float) * nz);  // csr values for matrix A
  x = (float *)malloc(sizeof(float) * N);
  rhs = (float *)malloc(sizeof(float) * N);
  diag = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    rhs[i] = 0.0;  // Initialize RHS
    x[i] = 0.0;    // Initial solution approximation
  }

  genLaplace(I, J, val, M, N, nz, rhs, diag, J_even, J_odd, I_even, I_odd, &idx_even, &idx_odd);

//   for (int i = 0; i < N; i++) printf("%0.f  ", diag[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N + 1; i++) printf("%d  ", I[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < nz; i++) printf("%d  ", J[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < nz; i++) printf("%.0f  ", val[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N; i++) printf("%.0f  ", rhs[i]);
//   printf("\n");
//   printf("\n");
// 
//   for (int i = 0; i < N; i++) {
//       int jj = I[i];
//       for (int j = 0; j < M; j++) {
// //           if ((j + i) % 2 == 0) {
//             if (J[jj] == j) {
//               printf("%.0f\t", val[jj]);
//               jj++;
//             } else { 
//               printf("%.0f\t", 0.);
//             }
// //            } else {
// //              if (J[jj] == j) {
// //                  jj++;
// //              }
// //              printf("x\t");
// //           }
//       }
//       printf("\n\n");
//   }

//   int ii = 0;
//      int jj = 0;
//   for (int i = 0; i < N; i++) {
//      for (int j = I[i]; j < I[i + 1]; j++) {
//         if((i + J[j]) % 2 == 0) {
//             printf("%d %d %.0f\n", i, J[j], val[j]);
//             J_even[jj] = J[j];
//             jj++;
//             ii++;
//         }
//         I_even[i] = ii;
//      }
//   } 

//   for (int i = 0; i < N + 1; i++) I_even[i] = (i % 2 == 1) ? I[i] + 1 : I[i]; 
//   for (int i = 0; i < N + 1; i++) I_odd[i] = (i % 2 == 0) ? I[i] + 1 : I[i]; 
// 
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N / 2; i++) printf("%d  ", I_even[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N / 2; i++) printf("%d  ", J_even[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N / 2; i++) printf("%d  ", I_odd[i]);
//   printf("\n");
//   printf("\n");
//   for (int i = 0; i < N / 2; i++) printf("%d  ", J_odd[i]);
//   printf("\n");
//   printf("\n");
// 
//   int jj_even = 0;
//   int jj_odd = 0;
//   for (int i = 0; i < N; i++) {
//       for (int j = 0; j < M; j++) {
//         if (J_even[jj_even] == j) {
//           printf("%.0f\t", val[I[i] + J_even[jj_even]]);
//           jj_even++;
//         } else if (J_odd[jj_odd] == j) { 
//           printf("[%.0f]\t", val[I[i] + J_odd[jj_odd]]);
//           jj_odd++;
//         } else if ((i + j) % 2 == 0){
//           printf("0\t");
//         } else {
//           printf("(0)\t");
//         }
//           
//       }
//       printf("\n\n");
//   }


  /* Create CUBLAS context */
  hipblasHandle_t cublasHandle = NULL;
  checkCudaErrors(hipblasCreate(&cublasHandle));

  /* Create CUSPARSE context */
  hipsparseHandle_t cusparseHandle = NULL;
  checkCudaErrors(hipsparseCreate(&cusparseHandle));

  /* Description of the A matrix */
  hipsparseMatDescr_t descr = 0;
  checkCudaErrors(hipsparseCreateMatDescr(&descr));
  checkCudaErrors(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

  /* Allocate required memory */
  checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_x_old, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_diff, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_p, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_diag, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_omega, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_valsILU0, nz * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_zm1, (N) * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_zm2, (N) * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_rm2, (N) * sizeof(float)));

  /* Wrap raw data into cuSPARSE generic API objects */
  hipsparseSpMatDescr_t matA = NULL;
  checkCudaErrors(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
  hipsparseDnVecDescr_t vecx = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_32F));
  hipsparseDnVecDescr_t vecxold = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecxold, N, d_x_old, HIP_R_32F));
  hipsparseDnVecDescr_t vecr = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecr, N, d_p, HIP_R_32F));
//   hipsparseDnMatDescr_t vecdiag = NULL;
//   checkCudaErrors(hipsparseCreateDnMat(&vecdiag, N, 1, N, d_diag, HIP_R_32F, HIPSPARSE_ORDER_ROW));

  /* Initialize problem data */
  checkCudaErrors(
      hipMemcpy(d_col, J, nz * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_val, val, nz * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_diag, diag, N * sizeof(float), hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemset(d_y, 0, sizeof(float) * N));

  /* Create ILU(0) info object */
  csrilu02Info_t infoILU = NULL;
  checkCudaErrors(hipsparseCreateCsrilu02Info(&infoILU));

  /* Create L factor descriptor and triangular solve info */
  hipsparseMatDescr_t descrL = NULL;
  checkCudaErrors(hipsparseCreateMatDescr(&descrL));
  checkCudaErrors(hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER));
  checkCudaErrors(hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT));
  csrsv2Info_t infoL = NULL;
  checkCudaErrors(hipsparseCreateCsrsv2Info(&infoL));

  /* Create U factor descriptor and triangular solve info */
  hipsparseMatDescr_t descrU = NULL;
  checkCudaErrors(hipsparseCreateMatDescr(&descrU));
  checkCudaErrors(hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER));
  checkCudaErrors(hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT));
  csrsv2Info_t infoU = NULL;
  checkCudaErrors(hipsparseCreateCsrsv2Info(&infoU));

  /* Allocate workspace for cuSPARSE */
  size_t bufferSize = 0;
  size_t tmp = 0;
  int stmp = 0;
  checkCudaErrors(hipsparseSpMV_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatminusone, matA, vecx,
      &floatzero, vecr, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &tmp));
  if (tmp > bufferSize) {
    bufferSize = stmp;
  }
  checkCudaErrors(hipsparseScsrilu02_bufferSize(
      cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU, &stmp));
  if (stmp > bufferSize) {
    bufferSize = stmp;
  }
  checkCudaErrors(hipsparseScsrsv2_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrL, d_val,
      d_row, d_col, infoL, &stmp));
  if (stmp > bufferSize) {
    bufferSize = stmp;
  }
  checkCudaErrors(hipsparseScsrsv2_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val,
      d_row, d_col, infoU, &stmp));
  if (stmp > bufferSize) {
    bufferSize = stmp;
  }
  checkCudaErrors(hipMalloc(&buffer, bufferSize));

  /* Conjugate gradient without preconditioning.
     ------------------------------------------

     Follows the description by Golub & Van Loan,
     "Matrix Computations 3rd ed.", Section 10.2.6  */
  printf("Convergence of Jacobi: \n");
  k = 0;
//   r0 = 0;

  // FIRST ITERATION
  checkCudaErrors(hipblasScopy(cublasHandle, N, d_x, 1, d_x_old, 1));
  checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1));

  // Ax{k}
  checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               &floatone, matA, vecxold, &floatzero, vecx, 
                               HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer)
                 );

  // (rhs - Ax{k})
  saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x, d_r, d_x); 

  // D^(-1)(rhs - Ax{k})
  checkCudaErrors(hipblasSsbmv(cublasHandle, HIPBLAS_FILL_MODE_LOWER, N, 0, 
                                &floatone, d_diag, 1, d_x, 1, &floatzero, d_x, 1));

//   x{k} + D^(-1)(rhs - Ax{k})
  checkCudaErrors(hipblasSaxpy(cublasHandle, N, &floatone, d_x_old, 1, d_x, 1));

  // residual
  saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x_old, d_x, d_diff); 
  checkCudaErrors(hipblasSdot(cublasHandle, N, d_diff, 1, d_diff, 1, &r1));

  std::swap(d_x_old, d_x);
  std::swap(vecxold, vecx);

//   hipsparseDnVecGetValues(vecxold, (void**)&d_p);
//   checkCudaErrors(
//       hipMemcpy(x, d_p, N * sizeof(float), hipMemcpyDeviceToHost));
//   for (int i = 0; i < N; i++) printf("%.2f ", x[i]);
//   printf("\n");

//   checkCudaErrors(
//       hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost));
// 
//   for (int i = 0; i < N; i++) printf("%.2f ", x[i]);
//   printf("\n");

  printf("%.2f %.2f\n", r1, tol*tol);

  while (r1 > tol * tol && k <= max_iter) {
    k++;

    // Ax{k}
    checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               &floatone, matA, vecxold, &floatzero, vecx, 
                               HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer)
                   );

//     if (k == 1) {
//       checkCudaErrors(
//           hipMemcpy(x, d_x_old, N * sizeof(float), hipMemcpyDeviceToHost));
// 
//       for (int i = 0; i < N; i++) printf("%.2f ", x[i]);
//       printf("\n");
//     }

    // (rhs - Ax{k})
    saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x, d_r, d_x); 

    // D^(-1)(rhs - Ax{k})
    checkCudaErrors(hipblasSsbmv(cublasHandle, HIPBLAS_FILL_MODE_LOWER, N, 0, 
                                &floatone, d_diag, 1, d_x, 1, &floatzero, d_x, 1));

//     x{k} + D^(-1)(rhs - Ax{k})
    checkCudaErrors(hipblasSaxpy(cublasHandle, N, &floatone, d_x_old, 1, d_x, 1));

    // residual
    saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x_old, d_x, d_diff); 
    checkCudaErrors(hipblasSdot(cublasHandle, N, d_diff, 1, d_diff, 1, &r1));

    std::swap(d_x_old, d_x);
    std::swap(vecxold, vecx);
  }

  printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));

  checkCudaErrors(
      hipMemcpy(x, d_x_old, N * sizeof(float), hipMemcpyDeviceToHost));

  /* check result */
  err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs[i]);

    if (diff > err) {
      err = diff;
    }
  }

  printf("  Convergence Test: %s \n", (k <= max_iter) ? "OK" : "FAIL");
  nErrors += (k > max_iter) ? 1 : 0;
  qaerr1 = err;

  if (0) {
    // output result in matlab-style array
    int n = (int)sqrt((double)N);
    printf("a = [  ");

    for (int iy = 0; iy < n; iy++) {
      for (int ix = 0; ix < n; ix++) {
        printf(" %f ", x[iy * n + ix]);
      }

      if (iy == n - 1) {
        printf(" ]");
      }

      printf("\n");
    }
  }

  /* Red-black Gauss-Seidel.
     ------------------------------------------
   */

  printf("Convergence of Jacobi: \n");
  k = 0;
//   r0 = 0;

  // FIRST ITERATION
  checkCudaErrors(hipblasScopy(cublasHandle, N, d_x, 1, d_x_old, 1));
  checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1));

  // Ax{k}
  checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               &floatone, matA, vecxold, &floatzero, vecx, 
                               HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer)
                 );

  cudaCheckErrors(hipsparseSbsrxmv(cusparseHandle, HIPSPARSE_DIRECTION_ROW,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, N / 2,
                                  M, N, nnz, &floatone, matA, val, mask_ptr,
                                  I_even, J_even, J, N, vecxold, &floatzero,
                                  vecx)
                 ); 

//   // (rhs - Ax{k})
//   saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x, d_r, d_x); 
// 
//   // D^(-1)(rhs - Ax{k})
//   checkCudaErrors(hipblasSsbmv(cublasHandle, HIPBLAS_FILL_MODE_LOWER, N, 0, 
//                                 &floatone, d_diag, 1, d_x, 1, &floatzero, d_x, 1));
// 
// //   x{k} + D^(-1)(rhs - Ax{k})
//   checkCudaErrors(hipblasSaxpy(cublasHandle, N, &floatone, d_x_old, 1, d_x, 1));
// 
//   // residual
//   saxpy<<<max(1. ,ceil(N / 1024)), 1024>>>(N, floatminusone, d_x_old, d_x, d_diff); 
//   checkCudaErrors(hipblasSdot(cublasHandle, N, d_diff, 1, d_diff, 1, &r1));
// 
//   std::swap(d_x_old, d_x);
//   std::swap(vecxold, vecx);

//   /* Preconditioned Conjugate Gradient using ILU.
//      --------------------------------------------
//      Follows the description by Golub & Van Loan,
//      "Matrix Computations 3rd ed.", Algorithm 10.3.1  */
// 
//   printf("\nConvergence of CG using ILU(0) preconditioning: \n");
// 
//   /* Perform analysis for ILU(0) */
//   checkCudaErrors(hipsparseScsrilu02_analysis(
//       cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU,
//       HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
// 
//   /* Copy A data to ILU(0) vals as input*/
//   checkCudaErrors(hipMemcpy(d_valsILU0, d_val, nz * sizeof(float),
//                              hipMemcpyDeviceToDevice));
// 
//   /* generate the ILU(0) factors */
//   checkCudaErrors(hipsparseScsrilu02(cusparseHandle, N, nz, descr, d_valsILU0,
//                                     d_row, d_col, infoILU,
//                                     HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
// 
//   /* perform triangular solve analysis */
//   checkCudaErrors(
//       hipsparseScsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                                N, nz, descrL, d_valsILU0, d_row, d_col, infoL,
//                                HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
//   checkCudaErrors(
//       hipsparseScsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                                N, nz, descrU, d_valsILU0, d_row, d_col, infoU,
//                                HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
// 
//   /* reset the initial guess of the solution to zero */
//   for (int i = 0; i < N; i++) {
//     x[i] = 0.0;
//   }
//   checkCudaErrors(
//       hipMemcpy(d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice));
//   checkCudaErrors(
//       hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
// 
//   k = 0;
//   checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));
// 
//   while (r1 > tol * tol && k <= max_iter) {
//     // preconditioner application: d_zm1 = U^-1 L^-1 d_r
//     checkCudaErrors(hipsparseScsrsv2_solve(
//         cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
//         descrL, d_valsILU0, d_row, d_col, infoL, d_r, d_y,
//         HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
//     checkCudaErrors(hipsparseScsrsv2_solve(
//         cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
//         descrU, d_valsILU0, d_row, d_col, infoU, d_y, d_zm1,
//         HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
// 
//     k++;
// 
//     if (k == 1) {
//       checkCudaErrors(hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1));
//     } else {
//       checkCudaErrors(
//           hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator));
//       checkCudaErrors(
//           hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator));
//       beta = numerator / denominator;
//       checkCudaErrors(hipblasSscal(cublasHandle, N, &beta, d_p, 1));
//       checkCudaErrors(
//           hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1));
//     }
// 
//     checkCudaErrors(hipsparseSpMV(
//         cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
//         &floatzero, vecomega, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
//     checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator));
//     checkCudaErrors(
//         hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator));
//     alpha = numerator / denominator;
//     checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1));
//     checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1));
//     checkCudaErrors(hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1));
//     nalpha = -alpha;
//     checkCudaErrors(hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1));
//     checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1));
//   }
// 
//   printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));
// 
//   checkCudaErrors(
//       hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost));
// 
//   /* check result */
//   err = 0.0;
// 
//   for (int i = 0; i < N; i++) {
//     rsum = 0.0;
// 
//     for (int j = I[i]; j < I[i + 1]; j++) {
//       rsum += val[j] * x[J[j]];
//     }
// 
//     diff = fabs(rsum - rhs[i]);
// 
//     if (diff > err) {
//       err = diff;
//     }
//   }
// 
//   printf("  Convergence Test: %s \n", (k <= max_iter) ? "OK" : "FAIL");
//   nErrors += (k > max_iter) ? 1 : 0;
//   qaerr2 = err;

  /* Destroy descriptors */
  checkCudaErrors(hipsparseDestroyCsrsv2Info(infoU));
  checkCudaErrors(hipsparseDestroyCsrsv2Info(infoL));
  checkCudaErrors(hipsparseDestroyCsrilu02Info(infoILU));
  checkCudaErrors(hipsparseDestroyMatDescr(descrL));
  checkCudaErrors(hipsparseDestroyMatDescr(descrU));
  checkCudaErrors(hipsparseDestroyMatDescr(descr));
  checkCudaErrors(hipsparseDestroySpMat(matA));
  checkCudaErrors(hipsparseDestroyDnVec(vecx));
  checkCudaErrors(hipsparseDestroyDnVec(vecr));

  /* Destroy contexts */
  checkCudaErrors(hipsparseDestroy(cusparseHandle));
  checkCudaErrors(hipblasDestroy(cublasHandle));

  /* Free device memory */
  free(I);
  free(J);
  free(val);
  free(x);
  free(rhs);
  checkCudaErrors(hipFree(buffer));
  checkCudaErrors(hipFree(d_col));
  checkCudaErrors(hipFree(d_row));
  checkCudaErrors(hipFree(d_val));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_x_old));
  checkCudaErrors(hipFree(d_diff));
  checkCudaErrors(hipFree(d_r));
  checkCudaErrors(hipFree(d_p));
  checkCudaErrors(hipFree(d_diag));
  checkCudaErrors(hipFree(d_omega));
  checkCudaErrors(hipFree(d_valsILU0));
  checkCudaErrors(hipFree(d_zm1));
  checkCudaErrors(hipFree(d_zm2));
  checkCudaErrors(hipFree(d_rm2));

  printf("\n");
  printf("Test Summary:\n");
  printf("   Counted total of %d errors\n", nErrors);
  printf("   qaerr1 = %f qaerr2 = %f\n\n", fabs(qaerr1), fabs(qaerr2));
  exit((nErrors == 0 && fabs(qaerr1) < 1e-5 && fabs(qaerr2) < 1e-5
            ? EXIT_SUCCESS
            : EXIT_FAILURE));
}
